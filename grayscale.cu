#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h>

#include "bitmap_image.hpp"

using namespace std;

__global__ void color_to_grey(uchar3 *input_image, uchar3 *output_image, int width, int height)
{
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    int i = (r * width) + c;

    if (r < height && c < width){
            
        float outputx = 0.0;
        outputx = (0.299f * input_image[i].x) + (0.578f * input_image[i].y) + (0.114f * input_image[i].z);

        unsigned char outputTotal = (unsigned char)outputx;
        
        output_image[i].x = outputTotal;    
        output_image[i].y = outputTotal;    
        output_image[i].z = outputTotal;    

        
    }

    // TODO: Convert color to grayscale by mapping components of uchar3 to RGB
    // x -> R; y -> G; z -> B
    // Apply the formula:
    // output = 0.299f * R + 0.578f * G + 0.114f * B
    // Hint: First create a mapping from 2D block and grid locations to an
    // absolute 2D location in the image then use that to calculate a 1D offset
}


int main(int argc, char **argv)
{
    if (argc != 2) {
        cerr << "format: " << argv[0] << " { 24-bit BMP Image Filename }" << endl;
        exit(1);
    }
    
    bitmap_image bmp(argv[1]);

    if(!bmp)
    {
        cerr << "Image not found" << endl;
        exit(1);
    }

    int height = bmp.height();
    int width = bmp.width();
    
    cout << "Image dimensions:" << endl;
    cout << "height: " << height << " width: " << width << endl;

    cout << "Converting " << argv[1] << " from color to grayscale..." << endl;

    //Transform image into vector of doubles
    vector<uchar3> input_image;
    rgb_t color;
    for(int x = 0; x < width; x++)
    {
        for(int y = 0; y < height; y++)
        {
            bmp.get_pixel(x, y, color);
            input_image.push_back( {color.red, color.green, color.blue} );
        }
    }

    vector<uchar3> output_image(input_image.size());

    uchar3 *d_in, *d_out;
    int img_size = (input_image.size() * sizeof(char) * 3);
    hipMalloc(&d_in, img_size);
    hipMalloc(&d_out, img_size);

    hipMemcpy(d_in, input_image.data(), img_size, hipMemcpyHostToDevice);
    hipMemcpy(d_out, input_image.data(), img_size, hipMemcpyHostToDevice);

    // TODO: Fill in the correnct blockSize and gridSize
    // currently only one block with one thread is being launched
    
    dim3 dimBlock(32, 32, 1);
    dim3 dimGrid(((int)ceil((float)width/dimBlock.x)), ((int)ceil((float)height/dimBlock.y)), 1);
    

    color_to_grey<<< dimGrid , dimBlock >>> (d_in, d_out, width, height);
    hipDeviceSynchronize();

    hipMemcpy(output_image.data(), d_out, img_size, hipMemcpyDeviceToHost);

    //Set updated pixels
    for(int x = 0; x < width; x++)
    {
        for(int y = 0; y < height; y++)
        {
            int pos = x * height + y;
            bmp.set_pixel(x, y, output_image[pos].x, output_image[pos].y, output_image[pos].z);
        }
    }

    cout << "Conversion complete." << endl;
    
    bmp.save_image("./grayscaled.bmp");

    hipFree(d_in);
    hipFree(d_out);
}